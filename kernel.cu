#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include "config.h"
#include "types.h"

u8 *d_virgin_bits;
u8 *d_trace_bits;
u8 *d_ret;
u8 *d_bitmap_changed;
u8 *cuda_ret;
u8 *cuda_bitmap_changed;

// Handle CUDA errors
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))
void HandleError(hipError_t err, const char *file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

extern "C" void setup_device_memory()
{
    const size_t size_ret = sizeof(u8);
    const size_t size_bitmap_changed = sizeof(u8);
    const size_t size_arrays = MAP_SIZE * sizeof(u8);

    HANDLE_ERROR(hipMalloc((void **)&d_virgin_bits, size_arrays));
    HANDLE_ERROR(hipMalloc((void **)&d_trace_bits,  size_arrays));
    HANDLE_ERROR(hipMalloc((void **)&d_bitmap_changed, size_bitmap_changed));
    HANDLE_ERROR(hipMalloc((void **)&d_ret, size_ret));

    cuda_ret = (u8*)malloc(size_ret);
    cuda_bitmap_changed = (u8*)malloc(size_bitmap_changed);
}

extern "C" void cleanup_device_memory()
{
    hipFree(d_virgin_bits);
    hipFree(d_trace_bits);
    hipFree(d_ret);
    hipFree(d_bitmap_changed);
    free(cuda_ret);
    free(cuda_bitmap_changed);
}

__global__ void has_new_bits_kernel(u8 *virgin_map, u8 *trace_bits, unsigned int *ret, unsigned int *bitmap_changed) {
    const u32 total_elements = MAP_SIZE >> 3;
    const u32 total_threads = blockDim.x * gridDim.x;
    const u32 elements_per_thread = total_elements / total_threads;
    u32 idx = threadIdx.x + blockIdx.x * blockDim.x;
    u64* current = ((u64*)trace_bits) + idx;
    u64* virgin = ((u64*)virgin_map) + idx;
    u8 local_ret = 0;

    for(u32 i=0; i < elements_per_thread; i++) {
        if (*current && (*current & *virgin)) {
            if (local_ret < 2) {
                u8* cur = (u8*)current;
                u8* vir = (u8*)virgin;

                if ((cur[0] && vir[0] == 0xff) || (cur[1] && vir[1] == 0xff) ||
                    (cur[2] && vir[2] == 0xff) || (cur[3] && vir[3] == 0xff) ||
                    (cur[4] && vir[4] == 0xff) || (cur[5] && vir[5] == 0xff) ||
                    (cur[6] && vir[6] == 0xff) || (cur[7] && vir[7] == 0xff)) 
                    local_ret = 2;
                else 
                    local_ret = 1;
            }

            *virgin &= ~*current;
        }

        current += total_threads;
        virgin += total_threads;
    }

    atomicMax(ret, (unsigned int)local_ret);

    if(threadIdx.x == 0) {
        *bitmap_changed = 1U;
    }
}

extern "C" u8 call_has_new_bits_kernel(u8 *virgin_map, u8 *trace_bits, u8 *afl_bitmap_changed)
{
    HANDLE_ERROR(hipMemcpy(d_virgin_bits, virgin_map, MAP_SIZE, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_trace_bits, trace_bits, MAP_SIZE, hipMemcpyHostToDevice));

    dim3 threads(256);
    dim3 blocks(960); 
    has_new_bits_kernel<<<blocks, threads>>>(d_virgin_bits, d_trace_bits, (unsigned int*)d_ret, (unsigned int*)d_bitmap_changed);

    HANDLE_ERROR(hipDeviceSynchronize());

    HANDLE_ERROR(hipMemcpy(virgin_map, d_virgin_bits, MAP_SIZE, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(cuda_ret, d_ret, sizeof(u8), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(cuda_bitmap_changed, d_bitmap_changed, sizeof(u8), hipMemcpyDeviceToHost));

    *afl_bitmap_changed = *cuda_bitmap_changed;

    return *cuda_ret;
}
